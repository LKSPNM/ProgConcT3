#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include "image.cu"
#define N_THREADS 8
#define N_BLOCKS 8

int readCharNum(char stop, FILE *fp){
	char *size = (char *) calloc(sizeof(char), 1);
        int i = 0;

        do{
                fread(&size[i], sizeof(char), 1, fp);
                size = (char *) realloc(size, sizeof(char)*(++i));
        }while(size[i - 1] != stop);
        size[i - 1] = '\0';

	int num = atoi(size);
	free(size);

	return num;
}

Image readImage(char *filename){
	Image img;
        FILE *fp = NULL;
	char trash;
	char filepath[60];
	unsigned char c;
	int i, j;

        //fp = fopen("./Images/sign_1.ppm", "rb+");
	sprintf(filepath, "./../Images/%s", filename);
	printf("%s\n", filepath);

	fp = fopen(filepath, "rb+");

        if(fp == NULL){ printf("File error!\n");	}

	//Read magic number(P6 or P3)
        printf("Start\n");
	fread(&img.mnum, sizeof(char), 2, fp);
	img.mnum[2] = '\0';
	printf("%s\n", img.mnum);
	fread(&trash, sizeof(char), 1, fp);

	//Read width
	img.column = readCharNum(((char) 32), fp);
	//Read height
	img.line = readCharNum('\n', fp);
	//Read maxcolor
	img.maxcolor = readCharNum('\n', fp);

	img.pixel = (Pixel **) malloc(sizeof(Pixel) * img.line);
	
	for(i = 0; i < img.line; ++i){ img.pixel[i] = (Pixel *) malloc(sizeof(Pixel) * img.column);	}

	if(img.mnum[0] == 'P' && img.mnum[1] == '6'){
		for(i = 0; i < img.line; ++i){
			for(j = 0; j < img.column; ++j){
				fread(&c, sizeof(unsigned char), 1, fp);
				img.pixel[i][j].R = c;
				fread(&c, sizeof(unsigned char), 1, fp);
                	        img.pixel[i][j].G = c;
				fread(&c, sizeof(unsigned char), 1, fp);
	                        img.pixel[i][j].B = c;
				//printf("[%d, %d]: R: %d\tG: %d\tB: %d\n", i, j, img.pixel[i][j].R, img.pixel[i][j].G, img.pixel[i][j].B);
			}
		}
	}else if(img.mnum[0] == 'P' && img.mnum[1] == '5'){
		for(i = 0; i < img.line; ++i){
                        for(j = 0; j < img.column; ++j){
                                img.pixel[i][j].R = 0;
                                fread(&c, sizeof(unsigned char), 1, fp);
                                img.pixel[i][j].G = c;
                                img.pixel[i][j].B = 0;
                                //printf("[%d, %d]: R: %d\tG: %d\tB: %d\n", i, j, img.pixel[i][j].R, img.pixel[i][j].G, img.pixel[i][j].B);
                        }
                }
	}
	//getchar();

	fclose(fp);

	return img;
}

int lenHelper(unsigned x) {
    if(x>=1000000000) return 10;
    if(x>=100000000) return 9;
    if(x>=10000000) return 8;
    if(x>=1000000) return 7;
    if(x>=100000) return 6;
    if(x>=10000) return 5;
    if(x>=1000) return 4;
    if(x>=100) return 3;
    if(x>=10) return 2;
    return 1;
}

void writeImage(char *filename, Image img, Pixel *newimg){
	FILE *fp;
	char eol = '\n';
	char filepath[60];
	char spc = ((char) 32);
	char ita[5];
	int i;

	//fp = fopen("./outImages/newsign_1.ppm", "wb+");
	sprintf(filepath, "./../outImages/out%s", filename);
	fp = fopen(filepath, "wb+");

	if(fp == NULL){	printf("File not open!\n"); return;	}

	printf("Start writing!\n");

	fwrite(img.mnum, sizeof(char), 2, fp);
	fwrite(&eol, sizeof(char), 1, fp);

	sprintf(ita, "%d", img.column);
        fwrite(ita, sizeof(char), lenHelper(img.column), fp);
	fwrite(&spc, sizeof(char), 1, fp);

	sprintf(ita, "%d", img.line);
	fwrite(ita, sizeof(char), lenHelper(img.line), fp);
	fwrite(&eol, sizeof(char), 1, fp);

	sprintf(ita, "%d", img.maxcolor);
	fwrite(ita, sizeof(char), lenHelper(img.maxcolor), fp);
	fwrite(&eol, sizeof(char), 1, fp);

	printf("writting pixels!\n");

	if(img.mnum[0] == 'P' && img.mnum[1] == '6'){
		for(i = 0; i < img.line * img.column; ++i){
				fwrite(&newimg[i].R, sizeof(unsigned char), 1, fp);
				fwrite(&newimg[i].G, sizeof(unsigned char), 1, fp);
				fwrite(&newimg[i].B, sizeof(unsigned char), 1, fp);
		}
	}else if(img.mnum[0] == 'P' && img.mnum[1] == '5'){
		for(i = 0; i < img.line; ++i){
                                fwrite(&newimg[i].G, sizeof(unsigned char), 1, fp);
                }
	}

	fclose(fp);

	printf("Finished!\n");

	return;
}

__global__ void smoothImage(int line, int column, Pixel *img, Pixel *newimg){
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	int i_new = (i * column) + j;

	//Calcula o novo valor do Pixel[i][j] verificando se o pixel esta no canto para colocar como 0 os pixels fora da imagem
	if(i == 0 && j == 0){
		newimg[i_new].R = (0 + 0 + 0 +
                                   0 + img[i_new].R + img[i_new + 1].R +
                                   0 + img[i_new + column].R + img[i_new + column + 1].R)
				/ 9;
				
		newimg[i_new].G = (0 + 0 + 0 +
                                   0 + img[i_new].G + img[i_new + 1].G +
                                   0 + img[i_new + column].G + img[i_new + column + 1].G)
				/9;
				
		newimg[i_new].B = (0 + 0 + 0 +
                		0 + img[i_new].B + img[i_new + 1].B +
                		0 + img[i_new + column].B + img[i_new + column + 1].B)
				/ 9;
	}else if(i == 0 && j >= column - 1){
                newimg[i_new].R = (0 + 0 + 0 +
                                   img[i_new - 1].R + img[i_new].R + 0 +
                                   img[i_new + column - 1].R + img[i_new + column].R + 0)
                                   / 9;

                                newimg[i_new].G = (0 + 0 + 0 +
                                                        img[i_new - 1].G + img[i_new].G + 0 +
                                                        img[i_new + column - 1].G + img[i_new + column].G + 0)
                                                        / 9;

                                newimg[i_new].B = (0 + 0 + 0 +
                                                        img[i_new - 1].B + img[i_new].B + 0 +
                                                        img[i_new + column - 1].B + img[i_new + column].B + 0)
                                                        / 9;

                        }else if(i >= line - 1 && j == 0){
                                newimg[i_new].R = (0 + img[i_new - column].R + img[i_new - column + 1].R +
                                                        0 + img[i_new].R + img[i_new + 1].R +
                                                        0 + 0 + 0)
                                                        / 9;

                                newimg[i_new].G = (0 + img[i_new - column].G + img[i_new - column + 1].G +
                                                        0 + img[i_new].G + img[i_new + 1].G +
                                                        0 + 0 + 0)
                                                        / 9;

                                newimg[i_new].B = (0 + img[i_new - column].B + img[i_new - column + 1].B +
                                                        0 + img[i_new].B + img[i_new + 1].B +
                                                        img[i_new].G + img[i_new].G + 0 +
                                                        0 + 0 + 0)
                                                        / 9;

                                newimg[i_new].B = (img[i_new - column - 1].B + img[i_new - column].B + 0 +
                                                        img[i_new - 1].B + img[i_new].B + 0 +
                                                        0 + 0 + 0)
                                                        / 9;
                        }else if(i == 0){
				newimg[i_new].R = (0 + 0 + 0 +
                                                        img[i_new - 1].R + img[i_new].R + img[i_new + 1].R +
                                                        img[i_new + column - 1].R + img[i_new + column].R + img[i_new + column + 1].R)
							/ 9;

				newimg[i_new].G = (0 + 0 + 0 +
                                                        img[i_new - 1].G + img[i_new].G + img[i_new + 1].G +
                                                        img[i_new + column - 1].G + img[i_new + column].G + img[i_new + column + 1].G)
                                                        / 9;

				
				newimg[i_new].B = (0 + 0 + 0 +
                                                        img[i_new - 1].B + img[i_new].B + img[i_new + 1].B +
                                                        img[i_new + column - 1].B + img[i_new + column].B + img[i_new + column + 1].B)
                                                        / 9;

			}else if(j == 0){
				newimg[i_new].R = (0 + img[i_new - column].R + img[i_new - column + 1].R+
                                                        0 + img[i_new].R + img[i_new + 1].R +
                                                        0 + img[i_new + column].R + img[i_new + column + 1].R)
							/ 9;
				
				newimg[i_new].G = (0 + img[i_new - column].G + img[i_new - column + 1].G+
                                                        0 + img[i_new].G + img[i_new + 1].G +
                                                        0 + img[i_new + column].G + img[i_new + column + 1].G)
                                                        / 9;

					
				newimg[i_new].B = (0 + img[i_new - column].B + img[i_new - column + 1].B+
                                                        0 + img[i_new].B + img[i_new + 1].B +
                                                        0 + img[i_new + column].B + img[i_new + column + 1].B)
                                                        / 9;
			}else if(i >= line - 1){
				newimg[i_new].R = (img[i_new - column - 1].R + img[i_new - column].R + img[i_new - column + 1].R+
                                                        img[i_new - 1].R + img[i_new].R + img[i_new + 1].R +
                                                        0 + 0 + 0)
                                                        / 9;


			newimg[i_new].G = (img[i_new - column - 1].G + img[i_new - column].G + img[i_new - column + 1].G+
                                                        img[i_new - 1].G + img[i_new].G + img[i_new + 1].G +
                                                        0 + 0 + 0)
                                                        / 9;

			newimg[i_new].B = (img[i_new - column - 1].B + img[i_new - column].B + img[i_new - column + 1].B+
                                                        img[i_new - 1].B + img[i_new].B + img[i_new + 1].B +
                                                        0 + 0 + 0)
                                                        / 9;
			}else if(j >= column - 1){
				newimg[i_new].R = (img[i_new - column - 1].R + img[i_new - column].R + 0 +
                                                        img[i_new - 1].R + img[i_new].R + 0 +
                                                        img[i_new + column - 1].R + img[i_new + column].R + 0)
                                                        / 9;

				newimg[i_new].G = (img[i_new - column - 1].G + img[i_new - column].G + 0 +
                                                        img[i_new - 1].G + img[i_new].G + 0 +
                                                        img[i_new + column - 1].G + img[i_new + column].G + 0)
                                                        / 9;


				newimg[i_new].B = (img[i_new - column - 1].B + img[i_new - column].B + 0 +
                                                        img[i_new - 1].B + img[i_new].B + 0 +
                                                        img[i_new + column - 1].B + img[i_new + column].B + 0)
                                                        / 9;

			}else{
				newimg[i_new].R = (img[i_new - column - 1].R + img[i_new - column].R + img[i_new - column + 1].R+
							img[i_new - 1].R + img[i_new].R + img[i_new + 1].R +
							img[i_new + column - 1].R + img[i_new + column].R + img[i_new + column + 1].R)
							/ 9;

				newimg[i_new].G = (img[i_new - column - 1].G + img[i_new - column].G + img[i_new - column + 1].G+
                                                        img[i_new - 1].G + img[i_new].G + img[i_new + 1].G +
                                                        img[i_new + column - 1].G + img[i_new + column].R + img[i_new + column + 1].G)
                                                        / 9;

				newimg[i_new].B = (img[i_new - column - 1].B + img[i_new - column].B + img[i_new - column + 1].B+
                                                        img[i_new - 1].B + img[i_new].B + img[i_new + 1].B +
                                                        img[i_new + column - 1].B + img[i_new + column].B + img[i_new + column + 1].B)
                                                        / 9;

	}

	__syncthreads();

	return;
}

Pixel *smoothInit(Image img){
	Pixel *newimg, *retimg, *piximg, *sntimg;
	int i, j;
	clock_t start, end;
        double cpu_time_used;

	retimg = (Pixel *) malloc(sizeof(Pixel) * img.line * img.column);
	piximg = (Pixel *) malloc(sizeof(Pixel) * img.line * img.column);
	for(i = 0; i < img.line; ++i){
		for(j = 0; j < img.column; ++j){
			piximg[i * (img.column) + j] = img.pixel[i][j]; 
		}
	}
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(img.column / threadsPerBlock.x, img.line / threadsPerBlock.y);

	hipMalloc((void **) &newimg, sizeof(Pixel) * img.line * img.column);
	hipMalloc((void **) &sntimg, sizeof(Pixel) * img.line * img.column);
	hipMemcpy(sntimg, piximg, sizeof(Pixel) * img.line * img.column, hipMemcpyHostToDevice);

	start = clock();
	smoothImage<<<numBlocks, threadsPerBlock>>>(img.line, img.column, sntimg, newimg);
	end = clock();

	cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

	printf("\n-----------------------------------\nTook %f seconds to execute \n-----------------------------------\n", cpu_time_used);

	hipMemcpy(retimg, newimg, sizeof(Pixel) * img.line * img.column, hipMemcpyDeviceToHost);

	hipFree(newimg);
	hipFree(sntimg);
	free(piximg);

	return retimg;
}


int main(){
	Image img;
	Pixel *newimg;
	char filename[60];
	int i;

	scanf("%s", filename);
	printf("%s\n", filename);

	img = readImage(filename);

	newimg = smoothInit(img);

	writeImage(filename, img, newimg);

	for(i = 0; i < img.line; ++i){	free(img.pixel[i]);	}
	free(img.pixel);

        free(newimg);

	return 0;
}
